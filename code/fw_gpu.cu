#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define GRAPH_SIZE 400

#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff

void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        int r;
        r = rand() % 40;
        if (r > 20) {
          r = INF;
        }

        D(i, j) = r;
      }
    }
  }
}

__global__ void gpu_calculate(int k, int graph_size, int *output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (D(i, k) + D(k, j) < D(i, j)) {
      D(i, j) = D(i, k) + D(k, j);
    }
}

void floyd_warshall_gpu(const int *graph, int graph_size, int *output) {
  dim3    blocks(GRAPH_SIZE/16,GRAPH_SIZE/16);
  dim3    threads(16,16);
  int *dev;
  int size = sizeof(int) * graph_size * graph_size;
  hipMalloc(&dev, size);
  hipMemcpy(dev, graph, size, hipMemcpyHostToDevice);
  for (int k = 0; k < graph_size; k++) {
    gpu_calculate<<<blocks, threads>>>(k, graph_size, dev);
  }
  //hipMemcpy(output, dev, size, hipMemcpyDeviceToHost);
  hipFree(dev);  
}

void floyd_warshall_cpu(const int *graph, int graph_size, int *output) {
  int i, j, k;

  memcpy(output, graph, sizeof(int) * graph_size * graph_size);
  if (memcmp(output graph, size) != 0) {
    fprintf(stderr, "FAIL!\n");
  } 

  /*for (int l = 0; l < 100; l++) {
    printf("cpu output:%d\n", output[l]);
  }*/

  for (k = 0; k < graph_size; k++) {
    for (i = 0; i < graph_size; i++) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
        }
      }
    }
  }
  /*for (int l = 0; l < 100; l++) {
    printf("cpu output:%d\n", output[l]);
  }*/
  if (memcmp(output graph, size) != 0) {
    fprintf(stderr, "FAIL!\n");
  } 
}

int main(int argc, char **argv) {
#define TIMER_START() gettimeofday(&tv1, NULL)
#define TIMER_STOP()                                                           \
  gettimeofday(&tv2, NULL);                                                    \
  timersub(&tv2, &tv1, &tv);                                                   \
  time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

  struct timeval tv1, tv2, tv;
  float time_delta;

  int *graph, *output_cpu, *output_gpu;
  int size;

  size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

  graph = (int *)malloc(size);
  assert(graph);

  output_cpu = (int *)malloc(size);
  assert(output_cpu);
  memset(output_cpu, 0, size);

  output_gpu = (int *)malloc(size);
  assert(output_gpu);

  generate_random_graph(graph, GRAPH_SIZE);

  fprintf(stderr, "running on cpu...\n");
  TIMER_START();
  floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  fprintf(stderr, "running on gpu...\n");
  TIMER_START();
  //floyd_warshall_gpu(graph, GRAPH_SIZE, output_gpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  if (memcmp(output_cpu, output_gpu, size) != 0) {
    fprintf(stderr, "FAIL!\n");
  } else {
    for (int k = 500; k < 1000; k++) {
      //printf("cpu:%d gpu:%d origin:%d\n", output_cpu[k], output_gpu[k], graph[k]);
    }
    
  }

  return 0;
}
