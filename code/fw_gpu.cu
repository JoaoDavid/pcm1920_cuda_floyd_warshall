#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define GRAPH_SIZE 2000

#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff

void generate_random_graph(int *output, int graph_size) {
  int i, j;
  int counter = 0;
  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        int r;
        r = rand() % 40;
        if (r > 20) {
          //r = INF;
        }

        D(i, j) = r;
        if(r == 0){
          counter++;
          D(i, j) = 1;
        }
      }
    }
  }
  printf("counter:%d\n", counter);
}

int gcd(int a, int b) { 
    if (b == 0) {
      return a; 
    }        
    return gcd(b, a % b);  
} 

__global__ void gpu_calculate(int k, int graph_size, int *output, int threads) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i < GRAPH_SIZE && j < GRAPH_SIZE){
      extern __shared__ int shared[];
      int* frozenZoneHoriz = &shared[0];
      int* frozenZoneVert = &shared[threads];
      /*if(threadIdx.y == 0){
        frozenZoneHoriz[threadIdx.x] = D(i, k);
      }
      if(threadIdx.x == 0){
        frozenZoneVert[threadIdx.y] = D(k, j);
      }*/
      if(threadIdx.x == threadIdx.y){
        frozenZoneHoriz[threadIdx.x] = D(i, k);
        frozenZoneVert[threadIdx.y] = D(k, j);
      }
      
      __syncthreads();

      if (frozenZoneHoriz[threadIdx.x] + frozenZoneVert[threadIdx.y] < D(i, j)) {
        D(i, j) = frozenZoneHoriz[threadIdx.x] + frozenZoneVert[threadIdx.y];
      }
    }
}

void floyd_warshall_gpu(const int *graph, int graph_size, int *output) {
  int threads = gcd(GRAPH_SIZE,32);
  /*if(threads == 1){
    double aux = GRAPH_SIZE / 16;
    threads = ceil(aux);
  }*/
  threads = 16;
  printf("threads per block %d x %d\n",threads,threads);
  dim3 threadsPerBlock(threads, threads);
  dim3 numBlocks(GRAPH_SIZE / threadsPerBlock.x, GRAPH_SIZE / threadsPerBlock.y); 
  int *dev;
  int size = sizeof(int) * graph_size * graph_size;
  hipMalloc(&dev, size);
  hipMemcpy(dev, graph, size, hipMemcpyHostToDevice);
  for (int k = 0; k < graph_size; k++) {
    gpu_calculate<<<numBlocks, threadsPerBlock, sizeof(int) * threads * 2>>>(k, graph_size, dev, threads);
  }
  hipMemcpy(output, dev, size, hipMemcpyDeviceToHost);
  hipFree(dev);  
}

void floyd_warshall_cpu(const int *graph, int graph_size, int *output) {
  int i, j, k;

  memcpy(output, graph, sizeof(int) * graph_size * graph_size);

  for (k = 0; k < graph_size; k++) {
    for (i = 0; i < graph_size; i++) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
        }
      }
    }
  }
}

int main(int argc, char **argv) {
#define TIMER_START() gettimeofday(&tv1, NULL)
#define TIMER_STOP()                                                           \
  gettimeofday(&tv2, NULL);                                                    \
  timersub(&tv2, &tv1, &tv);                                                   \
  time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

  struct timeval tv1, tv2, tv;
  float time_delta;

  int *graph, *output_cpu, *output_gpu;
  int size;

  size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

  graph = (int *)malloc(size);
  assert(graph);

  output_cpu = (int *)malloc(size);
  assert(output_cpu);
  memset(output_cpu, 0, size);

  output_gpu = (int *)malloc(size);
  assert(output_gpu);

  generate_random_graph(graph, GRAPH_SIZE);

  fprintf(stderr, "running on cpu...\n");
  TIMER_START();
  floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  fprintf(stderr, "running on gpu...\n");
  TIMER_START();
  floyd_warshall_gpu(graph, GRAPH_SIZE, output_gpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  if (memcmp(output_cpu, output_gpu, size) != 0) {
    fprintf(stderr, "FAIL!\n");
  } else {
    /*for (int k = 500; k < 550; k++) {
      printf("cpu:%d gpu:%d origin:%d\n", output_cpu[k], output_gpu[k], graph[k]);
    }*/
    printf("OK\n");
  }

  return 0;
}
